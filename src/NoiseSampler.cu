#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>

#include "cuda_noise.cuh"

#include "NoiseSampler.cuh"


__device__
float SampleNoise(float3 pos)
{
	int seed = 42;
	float scale = 0.01f;
	float result = cudaNoise::perlinNoise(pos, scale, seed);  // float3 pos, float scale, int seed

	scale = 0.06f;
	result += cudaNoise::perlinNoise(pos, scale, seed);

	scale = 0.12f;
	result += cudaNoise::perlinNoise(pos, scale, seed);
	return result;
}

__device__
float Waves(float3 p)
{
	//printf("density at (%f, %f, %f) is = %f\n", p.x, p.y, p.z, value);
	//std::cout << "position: " << p.x << std::endl;
	//printf("position %f %f %f\n", p.x, p.y, p.z);
	//return sin(p.x * 1.0) + cos(p.y * 1.0) + p.z - 2;

	//float value = sin(p.x * 1.0f) / 1.f + cos(p.y * 1.0f) / 1.f + p.z - 5.50f;
	//float value = sin(p.x * 0.5f) / 0.3f + p.y - 5.50f;
	float value = sin(p.x) + p.y - 6.0;

	//return sin(p.x) + cos(p.z) + p.y;
	return value;
}

__device__
void NoiseDensity(float* result, float3 position) {
  //float3 pos = make_float3(0.1, 0.1, 0.1);
  *result = SampleNoise(position);
}

__device__
void NoiseGradient(float3* result, float3 position) {
  //float3 pos = make_float3(0.1, 0.1, 0.1);
  //*result = sample_noise(pos);

	const float epsilon = 0.0001f;
	const float dx = SampleNoise(make_float3(position.x + epsilon, position.y, position.z)) - SampleNoise(make_float3(position.x - epsilon, position.y, position.z));
	const float dy = SampleNoise(make_float3(position.x, position.y + epsilon, position.z)) - SampleNoise(make_float3(position.x, position.y - epsilon, position.z));
	const float dz = SampleNoise(make_float3(position.x, position.y, position.z + epsilon)) - SampleNoise(make_float3(position.x, position.y, position.z - epsilon));

	//const float dy = SampleNoise(position + make_float3(0.f, epsilon, 0.f)) - SampleNoise(position - make_float3(0.f, epsilon, 0.f));
	//const float dz = SampleNoise(position + make_float3(0.f, 0.f, epsilon)) - SampleNoise(position - make_float3(0.f, 0.f, epsilon));

  *result = make_float3(dx, dy, dz);
  //return glm::normalize(glm::vec3(dx, dy, dz));
}

__device__
void Sample(float4* sample, float3 position)
{
	float density;
	NoiseDensity(&density, position);
	//density = Waves(position);

	float3 gradient;
	NoiseGradient(&gradient, position);

	*sample = make_float4(density, gradient.x, gradient.y, gradient.z);
}

__device__
float3 indexToPos(long idx, int size)
{
	//int idx = i + segmentStart;
	int x = idx % size;
	int y = (idx / size) % size;
	int z = idx / (size * size);
	return make_float3(x, y, z);
}
//samples[i] = Noise(min + glm::ivec3(x, y, z), noiseModule);

__global__
void CacheKernel(float4* results, float3 min, int size)
{
	unsigned long index = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride = blockDim.x * gridDim.x;

	int samplesPerDirection = 8 * size;
	unsigned long n = samplesPerDirection * samplesPerDirection * samplesPerDirection;
	for (int i = index; i < n; i += stride)
	{
		float3 offset = indexToPos(i, samplesPerDirection); // like this right?
		//float3 offset = indexToPos(i, size);

		// has to be float because sub-grid-point sampling for crossing points
		float3 position = make_float3(min.x + offset.x / 8.0, min.y + offset.y / 8.0, min.z + offset.z / 8.0);

		float4 sample;
		Sample(&sample, position);

		results[i] = sample;
	}
}

namespace CudaNoise
{
	void CacheArea(int minX, int minY, int minZ, int size, float4* cpuResults)
	{
		auto t0 = std::chrono::high_resolution_clock::now();
		float3 min = make_float3(minX, minY, minZ);
		float4* results;

		// 1 float for density, 3 floats for gradient, 
		// size^3 points in grid
		//8 samples per unit of grid (for figuring out zero crossing)
		unsigned int samplesPerDirection = 8 * size;
		unsigned long samples = samplesPerDirection * samplesPerDirection * samplesPerDirection;
		unsigned long dataSize = sizeof(float) * 4 * samples;

		int threadsPerBlock = 256;
		int numBlocks = (samples + threadsPerBlock - 1) / threadsPerBlock; // round up

		hipMallocManaged(&results, dataSize);
		auto t1 = std::chrono::high_resolution_clock::now();


		printf("Launching kernel\n");
		CacheKernel <<<numBlocks, threadsPerBlock >> > (results, min, size);
		auto t2 = std::chrono::high_resolution_clock::now();

		//printf("Syncing \n");
		//hipDeviceSynchronize();
		auto t3 = std::chrono::high_resolution_clock::now();

		printf("Copying to vector \n");
		hipMemcpy(cpuResults, results, dataSize, hipMemcpyDeviceToHost);
		auto t4 = std::chrono::high_resolution_clock::now();

		printf("Freeing memory \n");
		hipFree(results);
		auto t5 = std::chrono::high_resolution_clock::now();

		printf("Done \n");

		auto initializationTime = std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count();
		auto computeTime = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();
		auto syncTime = std::chrono::duration_cast<std::chrono::microseconds>(t3 - t2).count();
		auto memcpyTime = std::chrono::duration_cast<std::chrono::microseconds>(t4 - t3).count();
		auto freeTime = std::chrono::duration_cast<std::chrono::microseconds>(t5 - t4).count();
		std::cout << "init and malloc time " << initializationTime / 1000.f << "ms" << std::endl;
		std::cout << "compute time " << computeTime / 1000.f << "ms" << std::endl;
		std::cout << "synchronize time " << syncTime / 1000.f << "ms" << std::endl;
		std::cout << "memcpy time" << memcpyTime / 1000.f << "ms" << std::endl;
		std::cout << "hipFree time " << freeTime / 1000.f << "ms" << std::endl;
	}
}
