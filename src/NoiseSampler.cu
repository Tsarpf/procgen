#include "hip/hip_runtime.h"
#include <iostream>

#include "cuda_noise.cuh"

#include "NoiseSampler.cuh"

__device__
float SampleNoise(float3 pos)
{
  int seed = 42;
  float scale = 0.05f;
  return cudaNoise::perlinNoise(pos, scale, seed);  // float3 pos, float scale, int seed
}

__device__
float Waves(float3 p)
{
	//printf("density at (%f, %f, %f) is = %f\n", p.x, p.y, p.z, value);
	//std::cout << "position: " << p.x << std::endl;
	//printf("position %f %f %f\n", p.x, p.y, p.z);
	//return sin(p.x * 1.0) + cos(p.y * 1.0) + p.z - 2;

	//float value = sin(p.x * 1.0f) / 1.f + cos(p.y * 1.0f) / 1.f + p.z - 5.50f;
	float value = sin(p.x * 0.5f) / 0.3f + p.y - 5.50f;

	//return sin(p.x) + cos(p.z) + p.y;
	return value;
}

__device__
void NoiseDensity(float* result, float3 position) {
  //float3 pos = make_float3(0.1, 0.1, 0.1);
  *result = SampleNoise(position);
}

__device__
void NoiseGradient(float3* result, float3 position) {
  //float3 pos = make_float3(0.1, 0.1, 0.1);
  //*result = sample_noise(pos);

	const float epsilon = 0.0001f;
	const float dx = SampleNoise(make_float3(position.x + epsilon, position.y, position.z)) - SampleNoise(make_float3(position.x - epsilon, position.y, position.z));
	const float dy = SampleNoise(make_float3(position.x, position.y + epsilon, position.z)) - SampleNoise(make_float3(position.x, position.y - epsilon, position.z));
	const float dz = SampleNoise(make_float3(position.x, position.y, position.z + epsilon)) - SampleNoise(make_float3(position.x, position.y, position.z - epsilon));

	//const float dy = SampleNoise(position + make_float3(0.f, epsilon, 0.f)) - SampleNoise(position - make_float3(0.f, epsilon, 0.f));
	//const float dz = SampleNoise(position + make_float3(0.f, 0.f, epsilon)) - SampleNoise(position - make_float3(0.f, 0.f, epsilon));

  *result = make_float3(dx, dy, dz);
  //return glm::normalize(glm::vec3(dx, dy, dz));
}

__device__
void Sample(float4* sample, float3 position)
{
	float density;
	NoiseDensity(&density, position);
	//density = Waves(position);

	float3 gradient;
	NoiseGradient(&gradient, position);

	*sample = make_float4(density, gradient.x, gradient.y, gradient.z);
}

__device__
float3 indexToPos(long idx, int size)
{
	//int idx = i + segmentStart;
	int z = idx % size;
	int y = (idx / size) % size;
	int x = idx / (size * size);
	return make_float3(x, y, z);
}
//samples[i] = Noise(min + glm::ivec3(x, y, z), noiseModule);

__global__
void CacheKernel(float4* results, float3 min, int size)
{
	unsigned long index = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride = blockDim.x * gridDim.x;

	int samplesPerDirection = 8 * size;
	unsigned long n = samplesPerDirection * samplesPerDirection * samplesPerDirection;
	for (int i = index; i < n; i += stride)
	{
		float3 offset = indexToPos(i, samplesPerDirection); // like this right?
		//float3 offset = indexToPos(i, size);

		// has to be float because sub-grid-point sampling for crossing points
		float3 position = make_float3(min.x + offset.x / 8.0, min.y + offset.y / 8.0, min.z + offset.z / 8.0);

		float4 sample;
		Sample(&sample, position);

		results[i] = sample;
	}
}

namespace CudaNoise
{
	float4* CacheArea(int minX, int minY, int minZ, int size)
	{
		float3 min = make_float3(minX, minY, minZ);
		float4* results;

		// 1 float for density, 3 floats for gradient, 
		// size^3 points in grid
		//8 samples per unit of grid (for figuring out zero crossing)
		unsigned int samplesPerDirection = 8 * size;
		unsigned long samples = samplesPerDirection * samplesPerDirection * samplesPerDirection;
		unsigned long dataSize = sizeof(float) * 4 * samples;
		hipMallocManaged(&results, dataSize);

		int threadsPerBlock = 256;
		int numBlocks = (samples + threadsPerBlock - 1) / threadsPerBlock; // round up

		printf("Launching kernel\n");
		CacheKernel <<<numBlocks, threadsPerBlock >> > (results, min, size);

		printf("Syncing \n");
		hipDeviceSynchronize();

		printf("Done\n");
		return results;
	}
}
