#include "hip/hip_runtime.h"
#include <iostream>

#include "cuda_noise.cuh"

#include "NoiseSampler.cuh"

__global__ 
void test_kernel(float* result) {
  float3 pos = make_float3(0.1, 0.1, 0.1);
  *result = cudaNoise::perlinNoise(pos, 0.1, 42);  // float3 pos, float scale, int seed
}

namespace CudaNoise
{
  void Sample(void)
  {
    //int N = 3;
    //float3* result;
    float* result;

    printf("Malloc\n");
    hipMallocManaged(&result, sizeof(float));

    printf("Launching kernel\n");

    test_kernel <<<1, 1>>> (result);

    printf("Syncing \n");
    hipDeviceSynchronize();

    printf("Done\n");

    printf("Result %f\n", *result);
  }

  //std::vector<float> AsyncCache(glm::ivec3 min, int segmentStart, int sampleCount, int size)
  //{
  //  noise::module::Perlin noiseModule;
  //  std::vector<float> samples(sampleCount);

  //  for (int i = 0; i < sampleCount; i++)
  //  {
  //    int idx = i + segmentStart;
  //    int z = idx % size;
  //    int y = (idx / size) % size;
  //    int x = idx / (size * size);
  //    samples[i] = Noise(min + glm::ivec3(x, y, z), noiseModule);
  //  }
  //  return samples;
  //}

}
//   std::vector<std::vector<float>> BuildCache(const glm::ivec3 min, const unsigned size)
//   {
//     unsigned idxCount = size * size * size;
//     unsigned concurrentThreadsSupported = std::thread::hardware_concurrency();
// 
//     // perf-wise no sense in computing less than thousands of samples per thread, but for debugging it's nice
//     unsigned threads = std::min(concurrentThreadsSupported, idxCount / 8); 
// 
//     unsigned samplesPerSegment = idxCount / threads;
// 
//     unsigned extras = size % threads;
//     //assert(size % threads == 0);
// 
//     std::vector<std::future<std::vector<float>>> futureSamples;
//     std::vector<std::vector<float>> results;
//     for (unsigned i = 0; i < threads; i++)
//     {
//       int segmentStart = i * (idxCount / threads);
//       futureSamples.push_back(std::async(
//             std::launch::async,
//             AsyncCache,
//             min,
//             segmentStart,
//             i < threads-1 ? samplesPerSegment : samplesPerSegment + extras,
//             size));
//     }
// 
//     for (auto& future : futureSamples)
//     {
//       results.push_back(future.get());
//     }
// 
//     return results;
//   }
