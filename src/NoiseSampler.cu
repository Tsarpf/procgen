#include "hip/hip_runtime.h"
#include <iostream>

#include "cuda_noise.cuh"

#include "NoiseSampler.cuh"

__device__
float SampleNoise(float3 pos)
{
  int seed = 42;
  float scale = 0.5f;
  return cudaNoise::perlinNoise(pos, scale, seed);  // float3 pos, float scale, int seed
}

__global__ 
void test_kernel(float* result) {
  float3 pos = make_float3(0.1, 0.1, 0.1);
  *result = SampleNoise(pos);
}

__global__ 
void NoiseDensity(float* result, float3 position) {
  //float3 pos = make_float3(0.1, 0.1, 0.1);
  *result = SampleNoise(position);
}

__global__ 
void NoiseGradient(float3* result, float3 position) {
  //float3 pos = make_float3(0.1, 0.1, 0.1);
  //*result = sample_noise(pos);

	const float epsilon = 0.0001f;
	const float dx = SampleNoise(position + make_float3(epsilon, 0.f, 0.f)) - SampleNoise(position - make_float3(epsilon, 0.f, 0.f));
	const float dy = SampleNoise(position + make_float3(0.f, epsilon, 0.f)) - SampleNoise(position - make_float3(0.f, epsilon, 0.f));
	const float dz = SampleNoise(position + make_float3(0.f, 0.f, epsilon)) - SampleNoise(position - make_float3(0.f, 0.f, epsilon));

  *result = make_float3(dx, dy, dz);
  //return glm::normalize(glm::vec3(dx, dy, dz));
}

namespace CudaNoise
{
  void CacheArea()
  {
  }

  void CacheDensity(float* results, float3 min, int size)
  {

  }

  void CacheGradient(float* results, float3 min, int size)
  {
  }

  void Sample(void)
  {
    //int N = 3;
    //float3* result;
    float* result;
    //*result = 0.0f;

    printf("Malloc\n");
    hipMallocManaged(&result, sizeof(float));

    printf("Launching kernel\n");

    test_kernel <<<1, 1>>> (result);

    printf("Syncing \n");
    hipDeviceSynchronize();

    printf("Done\n");

    printf("Result %f\n", *result);
  }

  //std::vector<float> AsyncCache(glm::ivec3 min, int segmentStart, int sampleCount, int size)
  //{
  //  noise::module::Perlin noiseModule;
  //  std::vector<float> samples(sampleCount);

  //  for (int i = 0; i < sampleCount; i++)
  //  {
  //    int idx = i + segmentStart;
  //    int z = idx % size;
  //    int y = (idx / size) % size;
  //    int x = idx / (size * size);
  //    samples[i] = Noise(min + glm::ivec3(x, y, z), noiseModule);
  //  }
  //  return samples;
  //}

}
//   std::vector<std::vector<float>> BuildCache(const glm::ivec3 min, const unsigned size)
//   {
//     unsigned idxCount = size * size * size;
//     unsigned concurrentThreadsSupported = std::thread::hardware_concurrency();
// 
//     // perf-wise no sense in computing less than thousands of samples per thread, but for debugging it's nice
//     unsigned threads = std::min(concurrentThreadsSupported, idxCount / 8); 
// 
//     unsigned samplesPerSegment = idxCount / threads;
// 
//     unsigned extras = size % threads;
//     //assert(size % threads == 0);
// 
//     std::vector<std::future<std::vector<float>>> futureSamples;
//     std::vector<std::vector<float>> results;
//     for (unsigned i = 0; i < threads; i++)
//     {
//       int segmentStart = i * (idxCount / threads);
//       futureSamples.push_back(std::async(
//             std::launch::async,
//             AsyncCache,
//             min,
//             segmentStart,
//             i < threads-1 ? samplesPerSegment : samplesPerSegment + extras,
//             size));
//     }
// 
//     for (auto& future : futureSamples)
//     {
//       results.push_back(future.get());
//     }
// 
//     return results;
//   }
